
#include <hip/hip_runtime.h>
__global__ void GainAdjustKernel(int p_Width, int p_Height, float p_GainR, float p_GainG, float p_GainB, float p_GainA, const float* p_Input, float* p_Output)
{
   const int x = blockIdx.x * blockDim.x + threadIdx.x;
   const int y = blockIdx.y * blockDim.y + threadIdx.y;

   if ((x < p_Width) && (y < p_Height))
   {
       const int index = ((y * p_Width) + x) * 4;

       p_Output[index + 0] = p_Input[index + 0] * p_GainR;
       p_Output[index + 1] = p_Input[index + 1] * p_GainG;
       p_Output[index + 2] = p_Input[index + 2] * p_GainB;
       p_Output[index + 3] = p_Input[index + 3] * p_GainA;
   }
}

void RunCudaKernel(void* p_Stream, int p_Width, int p_Height, float* p_Gain, const float* p_Input, float* p_Output)
{
    dim3 threads(128, 1, 1);
    dim3 blocks(((p_Width + threads.x - 1) / threads.x), p_Height, 1);
    hipStream_t stream = static_cast<hipStream_t>(p_Stream);

    GainAdjustKernel<<<blocks, threads, 0, stream>>>(p_Width, p_Height, p_Gain[0], p_Gain[1], p_Gain[2], p_Gain[3], p_Input, p_Output);
}
